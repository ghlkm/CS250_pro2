
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdio.h>
#include <iostream>
#include <math.h>

#include <ctime>
#include <chrono>

__device__
inline int idx(int x, int y, int width) {
	return y*width+x;
}

__global__
void stencil_cuda(float* temp, float* temp2, float* conduct, int width, int height, int steps) {
	for ( int s = 0; s < steps; s++ ) {
		for ( int i = 1; i < width-1; i++ ) {
			for ( int j = 1; j < height-1; j++ ) {
				temp2[idx(i,j,width)] = 
					temp[idx(i,j,width)] +
					(
						(temp[idx(i-1,j,width)] 
						- temp[idx(i,j,width)]) *conduct[idx(i-1,j,width)]

						+ (temp[idx(i+1,j,width)] 
						- temp[idx(i,j,width)]) *conduct[idx(i+1,j,width)]


						+ (temp[idx(i,j-1,width)]
						- temp[idx(i,j,width)]) *conduct[idx(i,j-1,width)]
						+ (temp[idx(i,j+1,width)] 
						- temp[idx(i,j,width)]) *conduct[idx(i,j+1,width)]

					)*0.2;
			}
		}
		float* t = temp;
		temp = temp2;
		temp2 = t;
	}
}
void stencil_optimized(float* temp, float* conduct, int width, int height, int steps) {
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	printf( "Number of CUDA devices: %d\n", deviceCount );
	for ( int i = 0; i < deviceCount; i++ ) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf( "%d: %s\n", i, prop.name );
		printf( "Max %d threads per block\n", prop.maxThreadsPerBlock );
	}


	printf( "Starting cuda for %d steps on %dx%dx\n", steps, width, height );
	//cudaMemcpy(da, ha, N*sizeof(float), cudaMemcpyHostToDevice);
	float* dtemp, *dtemp2, *dconduct;
	hipMalloc(&dtemp, width*height*sizeof(float));
	hipMalloc(&dtemp2, width*height*sizeof(float));
	hipMalloc(&dconduct, width*height*sizeof(float));

	hipMemcpy(dtemp, temp, width*height*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dconduct, conduct, width*height*sizeof(float), hipMemcpyHostToDevice);

	stencil_cuda<<<1,1>>>(dtemp,dtemp2,dconduct,width,height,steps);
	printf( "Started kernel\n" );
	hipDeviceSynchronize();
	
	if ( steps % 2 == 0 ) {
		hipMemcpy(temp, dtemp, width*height*sizeof(float), hipMemcpyDeviceToHost);
	} else {
		hipMemcpy(temp, dtemp2, width*height*sizeof(float), hipMemcpyDeviceToHost);
	}
}
